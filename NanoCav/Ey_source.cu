#include "hip/hip_runtime.h"
// The format for p (the parameters) is [omega, phase_delay, alpha, delay].
Ey(i,j,k) += source(i,j,k) * sinf(p[0] * t + p[1]) * 
    expf(-p[2] * powf(t - p[3], 2.0));

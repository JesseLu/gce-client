#include "hip/hip_runtime.h"
float v = fmodf(t, 1.0) + kappa_inv;
subH(i + (dx), j + (dy), k + (dz)) = H_d0(i,j,k) + \
    (v/2.0) * (H(i,j,k) - H_d1(i,j,k)) + \
    (powf(v, 2.0)/2.0) * (H(i,j,k) - 2.0 * H_d0(i,j,k) + H_d1(i,j,k));

